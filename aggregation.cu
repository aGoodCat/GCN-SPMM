#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;
using Scalar = float;
using uint = unsigned int;
#define BLK_DIM 512
#define FeatureSize 128
class CSR{
public :
    uint *rowPtrs;
    uint *cols;
    Scalar *vals;
};
__global__ void Spmm(CSR sparse_adj, unsigned int *target_nodes, unsigned int *sampling_nodes, float * features, unsigned int M,
                     unsigned int sampling_size, unsigned int batch_size, float *output){
    unsigned int warp_id = threadIdx.x / 32;
    unsigned int lane_id = threadIdx.x % 32;
    unsigned int num_warps = blockDim.x / 32;
    unsigned int global_warp_id = blockIdx.x * num_warps + warp_id;
    __shared__ float shared_adj_vals[];
    __shared__ unsigned int shared_adj_cols[];
    __shared__ float temp_outcome[];
    for(unsigned int i=global_warp_id;i<batch_size;i+=gridDim.x * (blockDim.x / 32)){
        unsigned int target_node = target_nodes[i];
        for(unsigned int j=lane_id;j<sampling_size;j+=32){
            unsigned int sample_neighbor = sampling_nodes[i * sampling_size + j];
            shared_adj_vals[warp_id*sampling_size + j] = sparse_adj.vals[sparse_adj.rowPtrs[target_node] + sample_neighbor];
            shared_adj_cols[warp_id*sampling_size + j] = sparse_adj.cols[sparse_adj.rowPtrs[target_node] + sample_neighbor];
        }
        for(unsigned int j=lane_id;j<FeatureSize;j+=32){
            temp_outcome[warp_id*FeatureSize + j] = 0.0f;
        }
        __syncwarp();
        for(unsigned int k=0;k<sampling_size;k++){
            unsigned int neigh = shared_adj_cols[warp_id*sampling_size + k];
            float edge_val = shared_adj_vals[warp_id*sampling_size + k];
            for(unsigned int j=lane_id;j<FeatureSize;j+=32){
                float feature_val = features[neigh*FeatureSize + j];
                temp_outcome[warp_id*FeatureSize + j] += edge_val * feature_val;
            }
        }
        for(unsigned int j=lane_id;j<FeatureSize;j+=32){
            output[i * FeatureSize + j] = temp_outcome[warp_id*FeatureSize +j];
        }
    }
}